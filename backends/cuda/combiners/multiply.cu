#include "hip/hip_runtime.h"
#include "multiply.cuh"


__global__ void multiplyKernelF(float* output, float* input, const int width, const int height, const float factor) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height) {
        return;
    }
    output[(j * width) + i] = input[(j * width) + i] * factor;
}

__global__ void multiplyKernel(float* output, const float* in0, const float* in1, const int& width, const int& height) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= width || j >= height) {
        return;
    }

    output[i + (j * width)] = in0[i + (j * width)] * in1[i + (j * width)];
}

void cudaMultiplyLauncherF(float* output, float* input, const int width, const int height, float factor) {
#ifdef CUDA_KERNEL_TIMING
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

    // Setup dimensions of kernel launch using occupancy calculator.
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiplyKernel, 0, 0); //???
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    multiplyKernelF<<<grid, block>>>(output, input, width, height, factor);
    // Check for succesfull kernel launch
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    // Synchronize device
    err = hipDeviceSynchronize();
    cudaAssert(err);

#ifdef CUDA_KERNEL_TIMING
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

    // If this completes, kernel is done and "output" contains correct data.
}

void cudaMultiplyLauncher(float* out, const float* in0, const float* in1, const int& width, const int& height) {
    // Setup dimensions of kernel launch using occupancy calculator.
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiplyKernel, 0, 0); //???
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    multiplyKernel<<<grid, block>>>(output, in0, in1, width, height);
    // Check for succesfull kernel launch
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    // Synchronize device
    err = hipDeviceSynchronize();
    cudaAssert(err);
}