#include "hip/hip_runtime.h"
#include "min.cuh"

__global__ void MinKernel(float* output, const float* in0, const float* in1, const int width, const int height) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}
	float out_val = in0[(j * width) + i] < in1[(j * width) + i] ? in0[(j * width) + i] : in1[(j * width) + i];
	output[(j * width) + i] = out_val;
}

void MinLauncher(float* output, const float* in0, const float* in1, const int width, const int height) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	dim3 block(32, 32, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	MinKernel<<<grid, block>>>(output, in0, in1, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

}