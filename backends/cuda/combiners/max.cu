#include "hip/hip_runtime.h"
#include "max.cuh"

__global__ void MaxKernel(float *output, const float *in0, const float *in1, const int width, const int height) {
	const int i = blockDim.x * blockIdx.x * threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}

	float out_val = in0[(j * width) + i] > in1[(j * width) + i] ? in0[(j * width) + i] : in1[(j * width) + i];
	output[(j * width) + i] = out_val;
}

void MaxLauncher(float *output, const float *in0, const float *in1, const int width, const int height) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MaxKernel, 0, 0);
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	MaxKernel<<<grid, block>>>(output, in0, in1, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

}