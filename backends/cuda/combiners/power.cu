#include "hip/hip_runtime.h"
#include "power.cuh"
#include "..\..\cpp\modules\combiners\Power.h"


__global__ void powerKernel(float* output, float* input0, float* input1, const int width, const int height) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}
	float prev0, prev1;
	prev0 = input0[(j * width) + i];
	prev1 = input1[(j * width) + i];

	// Raise prev0 to the power of prev1 and write to the output.
	output[(j * width) + i] = powf(prev0, prev1);
}

void powerLauncher(float* output, float* input0, float* input1, const int width, const int height) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, powerKernel, 0, 0); 
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	powerKernel<<<grid, block >>>(output, input0, input1, width, height); //Call Kernel
	// Check for successful kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING
	// If this completes, kernel is done and "output" contains correct data.
}


