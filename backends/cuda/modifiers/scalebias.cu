#include "hip/hip_runtime.h"
#include "scalebias.cuh"

__global__ void scalebiasKernel(float* output, float* input, const int width, const int height, float scale, float bias) {
	// Get current pixel.
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}

	output[(j * width) + i] = (input[(j * width) + i] * scale) + bias; // for default value for scale is 1 and bias is 0;
}

void scalebiasLauncher(float* output, float* input, const int width, const int height, float scale, float bias){

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	dim3 block(32, 32, 1);
	dim3 grid((width - 1) / block.x + 1, (height - 1) / block.y + 1, 1);
	scalebiasKernel<<<grid, block>>>(output, input, width, height, scale, bias);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING
}