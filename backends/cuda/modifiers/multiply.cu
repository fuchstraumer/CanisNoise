#include "hip/hip_runtime.h"
#include "multiply.cuh"


__global__ void multiplyKernel(float* output, float* input, const int width, const int height, const float factor) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}
	output[(j * width) + i] = input[(j * width) + i] * factor;
}

void multiplyLauncher(float* output, float* input, const int width, const int height, float factor) {
#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiplyKernel, 0, 0); //???
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	multiplyKernel<<<grid, block>>>(output, input, width, height, factor);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

	// If this completes, kernel is done and "output" contains correct data.
}