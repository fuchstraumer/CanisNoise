#include "hip/hip_runtime.h"
#include "abs.cuh"
#include "..\..\cpp\modules\modifiers\Abs.h"

__global__ void absKernel(float* output, float* input, const int width, const int height) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}

	float prev = input[(width * j) + i];
	output[(j * width) + i] = (prev <= 0.0f) ? -prev : prev;
}

void absLauncher(float* output, float* input, const int width, const int height) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, absKernel, 0, 0); //???
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	absKernel<<<block,grid>>>(output, input, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

	// If this completes, kernel is done and "output" contains correct data.
}
