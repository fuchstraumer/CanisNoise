#include "hip/hip_runtime.h"
#include "combiners/add.cuh"

__global__ void AddKernel(float* output, float* input0, float* input1, const int width, const int height) {
    // Get current pixel.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= width || j >= height) {
        return;
    }
    // Reading from a surface still requires the byte offset, so multiply the x coordinate by the size of a float in bytes.
    // surf2Dread also writes the value at the point to a pre-existing variable, so declare soemthing like "prev" and pass
    // it as a reference (&prev) to the surf2Dread function.
    float prev0 = input0[(j * width) + i];
    float prev1 = input1[(j * width) + i];
    // Add values and store in output.
    output[(j * width) + i] = prev0 + prev1;
}

__global__ void AddKernel3D(float* output, float* input0, float* input1, const int width, const int height, const int depth) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int j = blockDim.y * blockIdx.y + threadIdx.y;
    const int k = blockDim.z * blockIdx.z + threadIdx.z;
    if (i >= width || j >= height || k >= depth) {
        return;
    }

    float prev0, prev1;
    prev0 = input0[i + (j * width) + (k * width * height)];
    prev1 = input1[i + (j * width) + (k * width * height)];

    output[i + (j * width) + (k * width * height)] = prev0 + prev1;
}

void cudaAddLauncher(float* output, float* input0, float* input1, const int width, const int height){
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, AddKernel, 0, 0); 
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    AddKernel<<<grid, block>>>(output, input0, input1, width, height);
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    err = hipDeviceSynchronize();
    cudaAssert(err);
}

void AddLauncher3D(float * output, float * input0, float * input1, const int width, const int height, const int depth){

}
