#include "hip/hip_runtime.h"
#include "combiners/max.cuh"

__global__ void MaxKernel(float *output, const float *in0, const float *in1, const int width, const int height) {
    const int i = blockDim.x * blockIdx.x * threadIdx.x;
    const int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= width || j >= height) {
        return;
    }

    float out_val = in0[(j * width) + i] > in1[(j * width) + i] ? in0[(j * width) + i] : in1[(j * width) + i];
    output[(j * width) + i] = out_val;
}

void cudaMaxLauncher(float *output, const float *in0, const float *in1, const int width, const int height) {
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MaxKernel, 0, 0);
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    MaxKernel<<<grid, block>>>(output, in0, in1, width, height);
    // Check for succesfull kernel launch
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    // Synchronize device
    err = hipDeviceSynchronize();
    cudaAssert(err);
}