#include "hip/hip_runtime.h"
#include "combiners/multiply.cuh"


__global__ void multiplyKernelF(float* output, float* input, const int width, const int height, const float factor) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height) {
        return;
    }
    output[(j * width) + i] = input[(j * width) + i] * factor;
}

__global__ void multiplyKernel(float* output, const float* in0, const float* in1, const int& width, const int& height) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= width || j >= height) {
        return;
    }

    output[i + (j * width)] = in0[i + (j * width)] * in1[i + (j * width)];
}

void cudaMultiplyLauncherF(float* output, float* input, const int width, const int height, float factor) {
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiplyKernelF, 0, 0); //???
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    multiplyKernelF<<<grid, block>>>(output, input, width, height, factor);
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    err = hipDeviceSynchronize();
    cudaAssert(err);
}

void cudaMultiplyLauncher(float* out, const float* in0, const float* in1, const int& width, const int& height) {
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiplyKernel, 0, 0); //???
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
    multiplyKernel<<<grid, block>>>(out, in0, in1, width, height);
    hipError_t err = hipGetLastError();
    cudaAssert(err);
    err = hipDeviceSynchronize();
    cudaAssert(err);
}